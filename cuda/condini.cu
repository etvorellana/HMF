
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>


/* Function prototypes */

float ran2(long *);


void condini(long n, long *idum, double p0, double theta0, double r[], double p[])
{

   long i;
   double lt,ptot;
 
   lt=0.0; 
   for (i=0;i<n;i++)
       {
       r[i]=((double) ran2(idum))*theta0;
       p[i]=(((double) ran2(idum))-0.5)*p0;
       lt+=p[i];
       };
   lt=lt/((double) n);

   ptot=0.0;
   for (i=0;i<n;i++)
       {
       p[i]=p[i]-lt;
       ptot+=p[i];     
       };

   printf("Total initial momentum: %lf\n",ptot);

   return;
}
